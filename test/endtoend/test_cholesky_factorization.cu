#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <>
#include <clblast.h>

#include "hipblas.h"
#include "hipsolver.h"



void checkError(size_t status){
  if(status != 0) {
    std::cout << "ERROR status non-zero: " << status << std::endl;
  }
}

//TODO: this won't work for huge values.
//we need this because floats don't have infinite precision.
bool floatRoughlyEquals(float a, float b){
  if(a==b){
    return true;
  }
  if(abs(a-b)<1){
    return true;
  }
}

bool arrayMatch(float* A, float* B, int N){
  for(int i=0; i<N; i++){
    if(!floatRoughlyEquals(A[i], B[i])){
      return false;
    }
  }
  return true;
}

int main(){
  std::cout << "Start test_cholesky_factorization." << std::endl;
  hipsolverHandle_t cusolverDnHandle;


  //The test:
  //Matrix_1 * Matrix_1 = Matrix_2
  //potrf(Matrix_2) == Matrix_1
  int N = 5;
  int leading_dimension_1 = N; //This is sometimes equal to the number of rows, it's not that complex is it? (haha).
  //It's the total number of rows of the matrix in memory.
  float host_Matrix_1[] =
  {1, 0, 0, 0, 0,
  1, 2, 0, 0, 0,
  1, 2, 3, 0, 0,
  56, 2, 9, 9, 0,
  56, 2, 9, 9, 2};






  hipblasHandle_t blas;
  hipblasCreate(&blas);
  hipblasSetStream(blas, stream);


  hipblasFillMode_t upperOrLower = HIPBLAS_FILL_MODE_LOWER;


  // Create hipStream_t, cuSolver and cuBlass handles
  int gpuid = 0;
  checkError(hipSetDevice(gpuid));
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, 0);

  hipblasHandle_t blas;
  checkError(hipblasCreate(&blas));
  hipblasSetStream(blas, stream);

  checkError(hipsolverDnCreate(&cusolverDnHandle));
  hipsolverSetStream(cusolverDnHandle, stream);


  //Move matrici to device
  hipDeviceptr_t device_Matrix_1;
  hipMalloc(&device_Matrix_1, N^2 * sizeof(float));
  hipMemcpyHtoDAsync(device_Matrix_1, host_Matrix_1, N^2 * sizeof(float), stream);



  hipDeviceptr_t device_Matrix_2;
  hipMalloc(&device_Matrix_2, N^2 * sizeof(float));
  float alpha = 1f;
  float beta = 1f;
  int leading_dimension_2 = leading_dimension_1;
  hipblasSgemm(blas, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, (float*) device_Matrix_1, leading_dimension_1, (float*) device_Matrix_1, &beta, device_Matrix_2, leading_dimension_1);




  int workspace_size;
  hipsolverDnSpotrf_bufferSize(cusolverDnHandle, upperOrLower, N, (float *) device_Matrix_2, leading_dimension_2, &workspace_size);
  hipDeviceptr_t workspace=0; //this is on the gpu
  hipMalloc(&workspace, workspace_size * sizeof(float));

  int devInfo;
  hipsolverDnSpotrf(cusolverDnHandle, upperOrLower, N, (float *) device_Matrix_2, leading_dimension_2, (float*) workspace, workspace_size, &devInfo);

  //We don't want an error do we?
  assert(devInfo==0);




  float host_Matrix_2[N^2];
  hipMemcpyDtoHAsync(host_Matrix_2, device_Matrix_2, N^2 * sizeof(float), stream);

  arrayMatch(host_Matrix_1, host_Matrix_2, N^2);

  /*for(int i=0; i<N; i++){
      for(int j=0; j<N; j++){
        std::cout << "Element(" << i << ", " << j << ") = " << host_Matrix_2[i*N+j] << std::endl;
      }
  }*/

  checkError(hipblasDestroy(blas));
  checkError(hipsolverDnDestroy(&cusolverDnHandle));
  hipStreamDestroy(stream);

  std::cout << "finished test_cholesky_factorization." << std::endl;
  return 0;
}
