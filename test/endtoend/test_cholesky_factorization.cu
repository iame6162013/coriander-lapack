#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include "hipblas.h"
#include "hipsolver.h"


void checkError(size_t status){
  if(status != 0) {
    std::cout << "ERROR status non-zero: " << status << std::endl;
  }
}

int main(){
  std::cout << "Start test_cholesky_factorization." << std::endl;
  hipsolverHandle_t cusolverDnHandle;


  //Add matrix, etc

  //below needs to be a Hermitian matrix, also positive-definite matrix?
  float host_Matrix_1[] = {1, 2, 3, 4};
  int numRows_1 = 2;
  hipblasFillMode_t upperOrLower = HIPBLAS_FILL_MODE_LOWER;


  // Create hipStream_t, cuSolver and cuBlass handles
  int gpuid = 0;
  checkError(hipSetDevice(gpuid));
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, 0);

  hipblasHandle_t blas;
  checkError(hipblasCreate(&blas));
  hipblasSetStream(blas, stream);

  checkError(hipsolverDnCreate(&cusolverDnHandle));
  hipsolverSetStream(cusolverDnHandle, stream);


  //Move matrici to device
  hipDeviceptr_t device_Matrix_1;
  hipMalloc(&device_Matrix_1, numRows_1^2 * sizeof(float));
  hipMemcpyHtoDAsync(device_Matrix_1, host_Matrix_1, numRows_1^2 * sizeof(float), stream);




  int leading_dimension_1 = numRows_1; //This is sometimes equal to the number of rows, it's complex (haha).
  //It's the total number of rows of the matrix in memory.
  int workspace_size;
  hipsolverDnSpotrf_bufferSize(cusolverDnHandle, upperOrLower, numRows_1, (float *) device_Matrix_1, leading_dimension_1, &workspace_size);
  hipDeviceptr_t workspace=0; //this is on the gpu

  //TODO: this is a hack, and this hack is absent from 'normal' cuda code. This needs to be fixed some way.
  if(workspace_size!=0){
  	hipMalloc(&workspace, workspace_size * sizeof(float));
  }

  int devInfo;
  hipsolverDnSpotrf(cusolverDnHandle, upperOrLower, numRows_1, (float *) device_Matrix_1, leading_dimension_1, (float*) workspace, workspace_size, &devInfo);

  //We don't want an error do we?
  assert(devInfo==0);

  //TODO; Check result.
  hipMemcpyDtoHAsync(host_Matrix_1, device_Matrix_1, numRows_1^2 * sizeof(float), stream);

  for(int i=0; i<numRows_1; i++){
      for(int j=0; j<numRows_1; j++){
        std::cout << "Element(" << i << ", " << j << ") = " << host_Matrix_1[i*numRows_1+j] << std::endl;
      }
  }

  checkError(hipblasDestroy(blas));
  checkError(hipsolverDnDestroy(&cusolverDnHandle));
  hipStreamDestroy(stream);

  std::cout << "finished test_cholesky_factorization." << std::endl;
  return 0;
}
