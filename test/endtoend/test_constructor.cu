#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include "hipsolver.h"

void checkError(size_t status){
  if(status != 0) {
    std::cout << "ERROR status non-zero: " << status << std::endl;
  }
}

int main(){
  std::cout << "Start test_constructor." << std::endl;
  hipsolverHandle_t cusolverDnHandle;

  // Create cuSolver handle
  int gpuid = 0;
  checkError(hipSetDevice(gpuid));
  checkError(hipsolverDnCreate(&cusolverDnHandle));
  checkError(hipsolverDnDestroy(&cusolverDnHandle));

  std::cout << "finished test_constructor." << std::endl;
  return 0;
}
