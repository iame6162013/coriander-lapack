#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include "hipsolver.h"

void checkError(size_t status){
  if(status != 0) {
    std::cout << "ERROR status non-zero: " << status << std::endl;
  }
}

int main(){
  cout << "Start test_constructor." << endl;
  hipsolverHandle_t cusolverDnHandle;

  // Create CUBLAS and CUDNN handles
  int gpuid = 0;
  checkError(hipSetDevice(gpuid));
  checkError(cusolverCreate(&cusolverDnHandle));
  checkError(hipsolverDnDestroy(&cusolverDnHandle));

  cout << "finished test_constructor." << endl;
  return 0;
}
